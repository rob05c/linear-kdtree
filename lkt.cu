#include "hip/hip_runtime.h"
#include "lkt.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>    
#include <linux/hip/hip_runtime.h>

/// \returns the device totalGlobalMem
inline size_t GetDeviceMemory() {
  hipDeviceProp_t properties;
  int deviceNum;
  hipGetDevice(&deviceNum);
  hipGetDeviceProperties(&properties, deviceNum);
  return properties.totalGlobalMem;
}

__global__ void create_mortoncodes_kernel(lkt_point* points, mortoncode_t* codes, const fixlentree<lkt_split_point>::node* splitpoints, size_t len) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i >= len)
    return; // skip the final block remainder

  mortoncode_t& code = codes[i];
  code = 0;

  const lkt_point& point = points[i];

  bool is_x = true;
  size_t code_i = 0;
  for(index_t j = 0; j != fixlentree<lkt_split_point>::tree_end;) {
    const lkt_split_point& splitpoint = splitpoints[j].value;

    const int left = is_x * (point.x < splitpoint.value) + !is_x * (point.y < splitpoint.value);

    code = code | (left << code_i);

    j = splitpoints[j].left * left + splitpoints[j].right * !left;
    is_x = !is_x;
    ++code_i;
  }
}

/// \return array of morton codes, of len length. Caller takes ownership.
mortoncode_t* lkt_create_mortoncodes_parallel(lkt_point* points, size_t len, const fixlentree<lkt_split_point>::node* splitpoints) {
  const size_t THREADS_PER_BLOCK = 512;

  lkt_point*                         cuda_points;
  mortoncode_t*                      cuda_codes;
  fixlentree<lkt_split_point>::node* cuda_splitpoints;

  hipMalloc((void**)&cuda_points,      len * sizeof(lkt_point));
  hipMalloc((void**)&cuda_codes,       len * sizeof(mortoncode_t));
  hipMalloc((void**)&cuda_splitpoints, len * sizeof(fixlentree<lkt_split_point>::node));
  hipMemcpy(cuda_points,      points,      len * sizeof(lkt_point),                         hipMemcpyHostToDevice);
  hipMemcpy(cuda_splitpoints, splitpoints, len * sizeof(fixlentree<lkt_split_point>::node), hipMemcpyHostToDevice);

  create_mortoncodes_kernel<<<(len + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(cuda_points, cuda_codes, cuda_splitpoints, len);

  mortoncode_t* codes = new mortoncode_t[len];
  hipMemcpy(codes, cuda_codes, len * sizeof(mortoncode_t), hipMemcpyDeviceToHost);
  hipFree(cuda_points);
  hipFree(cuda_codes);
  hipFree(cuda_splitpoints);
  return codes;
}
